#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// Definición de la función f(x)
_host_ _device_ double f(double x){
  return x*x;
}

// Implementación de la función atomicAdd para números de punto flotante de doble precisión
_device_ double atomicAddDouble(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

// Kernel para calcular la integral utilizando CUDA
_global_ void integrate(double a, double b, int n, double h, double* result) {
  int i;
  double x, sum = 0.0;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for (i = idx + 1; i < n; i += stride) {
    x = a + i * h;
    sum += f(x);
  }
  
  sum *= 2.0;
  
  atomicAddDouble(result, sum);
}

// Función para obtener el tiempo actual en segundos con precisión de microsegundos
double getCurrentTime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}

int main(){
  int n, blockSize, numBlocks;
  double a, b, h, integral;
  double* result;
  double* dev_result;
  
  // Solicitar al usuario la entrada necesaria
  printf("\nIngrese el número de subintervalos: ");
  scanf("%d", &n);
  printf("\nIngrese el límite inicial: ");
  scanf("%lf", &a);
  printf("\nIngrese el límite final: ");
  scanf("%lf", &b);
  
  // Calcular el tamaño del intervalo
  h = fabs(b - a) / n;
  
  // Configurar los bloques y los hilos
  blockSize = 256; // Puedes ajustar el tamaño del bloque según tus necesidades
  numBlocks = (n + blockSize - 1) / blockSize;
  
  // Asignar memoria para el resultado en el host y en el device
  result = (double*)malloc(sizeof(double));
  hipMalloc((void**)&dev_result, sizeof(double));
  
  // Inicializar el resultado en 0
  *result = 0.0;
  hipMemcpy(dev_result, result, sizeof(double), hipMemcpyHostToDevice);
  
  // Medir el tiempo de inicio
  double start = getCurrentTime();
  
  // Lanzar el kernel en paralelo
  integrate<<<numBlocks, blockSize>>>(a, b, n, h, dev_result);
  hipDeviceSynchronize();
  
  // Medir el tiempo de fin
  double end = getCurrentTime();
  
  // Copiar el resultado de vuelta al host
  hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost);
  
  // Calcular el resultado de la integral
  integral = (h / 2) * (f(a) + f(b) + *result);
  
  // Imprimir la respuesta
  printf("\nEl resultado de la integral es: %lf\n", integral);
  
  // Calcular el tiempo transcurrido
  double elapsedTime = end - start;
  printf("Tiempo transcurrido: %.6f segundos\n", elapsedTime);
  
  // Cálculo del speedup y la escalabilidad
  int numThreads = numBlocks * blockSize;
  double sequentialTime = integral;
  double parallelTime = elapsedTime;
  double speedup = sequentialTime / parallelTime;
  double scalability = sequentialTime / (parallelTime * numThreads);
  
  printf("Speedup: %.2f\n", speedup);
  printf("Escalabilidad: %.2f\n", scalability);
  
  // Nueva métrica: Throughput
  double throughput = integral / elapsedTime;
  printf("Throughput: %.2f unidades por segundo\n", throughput);
  
  // Liberar memoria
  free(result);
  hipFree(dev_result);
  
  return 0;
}
